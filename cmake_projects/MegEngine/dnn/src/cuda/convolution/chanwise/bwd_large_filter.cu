#include "hip/hip_runtime.h"
#include "./kern.cuh"
#include "./kern_helper.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
#include "src/cuda/convolution/chanwise/launch_config.cuh"
#include "src/cuda/fp16_help.cuh"

using namespace megdnn;
using namespace cuda;
using namespace convolution;
using namespace chanwise;

#include "src/cuda/conv_bias/chanwise/depthwise_large_filter_algo.cuh"

namespace megdnn {
namespace cuda {
namespace convolution {
namespace chanwise {

// =====================================fwd=====================================

template <>
void run_bwd_depthwise_large_filter(
        float* dst, const float* src, const float* flt, const Param& param,
        hipStream_t stream) {
    INSTANCE(float, float2, DepthwiseConv2dDirection::DIRECTION_BACKWARD)
}

#if CUDA_VERSION >= 9000
template <>
void run_bwd_depthwise_large_filter(
        __half* dst, const __half* src, const __half* flt, const Param& param,
        hipStream_t stream) {
    INSTANCE(__half, __half2, DepthwiseConv2dDirection::DIRECTION_BACKWARD)
}
#endif

}  // namespace chanwise
}  // namespace convolution
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cuda.doxygen
