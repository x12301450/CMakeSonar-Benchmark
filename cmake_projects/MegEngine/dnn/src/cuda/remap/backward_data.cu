#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "src/common/rounding_converter.cuh"
#include "src/cuda/cv/kernel_common.cuh"
#include "src/cuda/remap/common.h"
#include "src/cuda/utils.cuh"

using namespace megdnn;
using namespace cuda;
using namespace remap;
using namespace rounding;

namespace {

template <const uint32_t format>
__device__ inline int get_offset(
        int height, int width, int channel, int h, int w, int c);

template <>
__device__ inline int get_offset<param_enumv::Remap::Format::NCHW>(
        int height, int width, int channel, int h, int w, int c) {
    return channel * h * w + height * w + width;
}

template <typename ctype, const uint32_t format, ::BorderMode bmode>
struct GetSrcData {
    __device__ static inline int get_index(
            int height, int width, int channel, int h, int w, int c) {
        height = megcv::border_interpolate<bmode>(height, h);
        width = megcv::border_interpolate<bmode>(width, w);
        return get_offset<format>(height, width, channel, h, w, c);
    }
};

template <typename ctype, const uint32_t format>
struct GetSrcData<ctype, format, ::BorderMode::BORDER_CONSTANT> {
    __device__ static inline int get_index(
            int height, int width, int channel, int h, int w, int c) {
        return (height >= 0 && height < h && width >= 0 && width < w)
                     ? get_offset<format>(height, width, channel, h, w, c)
                     : -1;
    }
};

__device__ inline float round_half_to_even(float f) {
    const float round_away_from_zero = round(f);
    const float diff = round_away_from_zero - f;

    if ((diff != 0.5f) && (diff != -0.5f)) {
        return round_away_from_zero;
    }

    if (fmod(round_away_from_zero, 2.0f) == 0.0f) {
        return round_away_from_zero;
    }

    return f - diff;
}

template <typename ctype, const uint32_t format, ::BorderMode bmode>
__global__ void kern_general_nearest(
        ctype* __restrict grad, const float* map_xy, const ctype* diff, int C, int IH,
        int IW, int OH, int OW) {
    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;
    grad += blockIdx.z * C * IH * IW;
    diff += blockIdx.z * C * OH * OW;
    map_xy += blockIdx.z * 2 * OH * OW;

    if (ow < OW && oh < OH) {
        float index_col = map_xy[oh * OW * 2 + ow * 2 + 0];
        float index_row = map_xy[oh * OW * 2 + ow * 2 + 1];
        int col = static_cast<int>(round_half_to_even(index_col));
        int row = static_cast<int>(round_half_to_even(index_row));
        for (int c = 0; c < C; ++c) {
            ctype hidden = diff[get_offset<format>(oh, ow, c, OH, OW, C)];
            int idx =
                    GetSrcData<ctype, format, bmode>::get_index(row, col, c, IH, IW, C);
            if (idx != -1) {
                atomic_add(grad + idx, hidden);
            }
        }
    }
}

template <typename ctype, const uint32_t format, ::BorderMode bmode>
__global__ void kern_general_linear(
        ctype* __restrict grad, const float* map_xy, const ctype* diff, int C, int IH,
        int IW, int OH, int OW) {
    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;
    grad += blockIdx.z * C * IH * IW;
    diff += blockIdx.z * C * OH * OW;
    map_xy += blockIdx.z * 2 * OH * OW;
    RoundingConverter<ctype> round_converter;

    if (ow < OW && oh < OH) {
        float index_col = map_xy[oh * OW * 2 + ow * 2 + 0];
        float index_row = map_xy[oh * OW * 2 + ow * 2 + 1];
        int col = static_cast<int>(floor(index_col));
        int row = static_cast<int>(floor(index_row));
        float v = index_col - col;  // alphah
        float u = index_row - row;  // alphaw
        const float one = 1.f;
        for (int c = 0; c < C; ++c) {
            float hidden =
                    static_cast<float>(diff[get_offset<format>(oh, ow, c, OH, OW, C)]);

            int a00 = GetSrcData<ctype, format, bmode>::get_index(
                    row + 0, col + 0, c, IH, IW, C);
            if (a00 != -1) {
                atomic_add(grad + a00, round_converter((one - u) * (one - v) * hidden));
            }

            int a01 = GetSrcData<ctype, format, bmode>::get_index(
                    row + 0, col + 1, c, IH, IW, C);
            if (a01 != -1) {
                atomic_add(grad + a01, round_converter((one - u) * v * hidden));
            }

            int a10 = GetSrcData<ctype, format, bmode>::get_index(
                    row + 1, col + 0, c, IH, IW, C);
            if (a10 != -1) {
                atomic_add(grad + a10, round_converter(u * (one - v) * hidden));
            }

            int a11 = GetSrcData<ctype, format, bmode>::get_index(
                    row + 1, col + 1, c, IH, IW, C);
            if (a11 != -1) {
                atomic_add(grad + a11, round_converter(u * v * hidden));
            }
        }
    }
}

template <
        typename ctype, const uint32_t format, ::BorderMode bmode,
        ::InterpolationMode imode>
void dispatch_backwarddata(
        ctype* grad, const float* map_xy, const ctype* diff, int N, int C, int IH,
        int IW, int OH, int OW, hipStream_t stream) {
    const int BX = 32, BY = 16;
    const int max_batch_size = 65535;
    while (N) {
        size_t curr_batch_size = N < max_batch_size ? N : max_batch_size;
        dim3 threads(BX, BY);
        dim3 blocks((OW + BX - 1) / BX, (OH + BY - 1) / BY, curr_batch_size);

        cuda_check(hipMemsetAsync(
                grad, 0, sizeof(ctype) * curr_batch_size * C * IH * IW, stream));
        if (imode == ::InterpolationMode::INTER_NEAREST) {
            kern_general_nearest<ctype, format, bmode><<<blocks, threads, 0, stream>>>(
                    grad, map_xy, diff, C, IH, IW, OH, OW);
        } else if (imode == ::InterpolationMode::INTER_LINEAR) {
            kern_general_linear<ctype, format, bmode><<<blocks, threads, 0, stream>>>(
                    grad, map_xy, diff, C, IH, IW, OH, OW);
        }

        N -= curr_batch_size;
        grad += curr_batch_size * C * IH * IW;
        diff += curr_batch_size * C * OH * OW;
        map_xy += curr_batch_size * 2 * OH * OW;
    }
}

}  // anonymous namespace

namespace megdnn {
namespace cuda {
namespace remap {

template <
        typename ctype, const uint32_t format, ::BorderMode bmode,
        ::InterpolationMode imode>
void backwarddata_proxy(
        ctype* grad, const float* map_xy, const ctype* diff, int N, int C, int IH,
        int IW, int OH, int OW, hipStream_t stream) {
    dispatch_backwarddata<ctype, format, bmode, imode>(
            grad, map_xy, diff, N, C, IH, IW, OH, OW, stream);
    after_kernel_launch();
}

#define INST(ctype, format, bmode, imode)                                     \
    template void backwarddata_proxy<                                         \
            ctype, param_enumv::Remap::Format::format, ::BorderMode::bmode,   \
            ::InterpolationMode::imode>(                                      \
            ctype*, const float*, const ctype*, int, int, int, int, int, int, \
            hipStream_t);

#define FOR_FORMAT_BMODE(ctype)                          \
    INST(ctype, NCHW, BORDER_CONSTANT, INTER_NEAREST)    \
    INST(ctype, NCHW, BORDER_REPLICATE, INTER_NEAREST)   \
    INST(ctype, NCHW, BORDER_REFLECT, INTER_NEAREST)     \
    INST(ctype, NCHW, BORDER_REFLECT_101, INTER_NEAREST) \
    INST(ctype, NCHW, BORDER_WRAP, INTER_NEAREST)        \
    INST(ctype, NCHW, BORDER_CONSTANT, INTER_LINEAR)     \
    INST(ctype, NCHW, BORDER_REPLICATE, INTER_LINEAR)    \
    INST(ctype, NCHW, BORDER_REFLECT, INTER_LINEAR)      \
    INST(ctype, NCHW, BORDER_REFLECT_101, INTER_LINEAR)  \
    INST(ctype, NCHW, BORDER_WRAP, INTER_LINEAR)

FOR_FORMAT_BMODE(float)
DNN_INC_FLOAT16(FOR_FORMAT_BMODE(dt_bfloat16))
DNN_INC_FLOAT16(FOR_FORMAT_BMODE(dt_float16))

#undef FOR_FORMAT_BMODE
#undef INST

}  // namespace remap
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cpp.doxygen
