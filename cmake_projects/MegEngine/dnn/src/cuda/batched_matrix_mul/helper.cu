#include "hip/hip_runtime.h"
#include "src/cuda/batched_matrix_mul/helper.cuh"

namespace {

template <typename T>
__global__ void kernel(T* Xs, T start, uint32_t step, uint32_t n) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        Xs[i] = start + i * step;
    }
}

}  // anonymous namespace

namespace megdnn {
namespace cuda {
namespace batched_matrix_mul {

template <typename T>
void arange(T* Xs, T start, uint32_t step, uint32_t n, hipStream_t stream) {
    uint32_t threads = NR_THREADS;
    uint32_t blocks = DIVUP(n, threads);
    kernel<T><<<blocks, threads, 0, stream>>>(Xs, start, step, n);
    after_kernel_launch();
}

template void arange<uintptr_t>(
        uintptr_t*, uintptr_t, uint32_t, uint32_t, hipStream_t);

}  // namespace batched_matrix_mul
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cpp.doxygen
