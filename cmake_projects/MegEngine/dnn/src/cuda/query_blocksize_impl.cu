#include "src/cuda/kernel_common/diagnostic_prologue.cuh"
#include "src/cuda/query_blocksize.cuh"
#include "src/cuda/utils.cuh"

using namespace megdnn;
using namespace cuda;

/*
 * Note: hipOccupancyMaxPotentialBlockSizeVariableSMem is only available when
 * compiled by nvcc, but it is implemented as a __host__ __device__ function. So
 * we implement a device wrapper
 */
namespace {

struct SmemGetterWrapper {
    SmemGetter getter;

    __device__ __host__ int operator()(int block_size) const {
#if __CUDA_ARCH__
        // device func should never be called
        int* ptr = 0;
        *ptr = 23;
#else
        if (getter.func) {
            return getter.func(block_size, getter.user_data);
        }
#endif
        return 0;
    }
};

}  // anonymous namespace

LaunchConfig cuda::detail::query_launch_config_for_kernel_uncached(
        const void* kern, const SmemGetter& smem) {
    SmemGetterWrapper s;
    s.getter = smem;
    LaunchConfig ret;
    cuda_check(hipOccupancyMaxPotentialBlockSizeVariableSMem(
            &ret.grid_size, &ret.block_size, kern, s));
    return ret;
}

#include "src/cuda/kernel_common/diagnostic_epilogue.cuh"
// vim: ft=cpp syntax=cpp.doxygen foldmethod=marker foldmarker=f{{{,f}}}
