#include "hip/hip_runtime.h"
#include "megdnn/dtype.h"
#include "src/cuda/eye/eye.cuh"
#include "src/cuda/utils.cuh"

namespace {

template <typename T>
__global__ void kernel(T* dst, uint32_t m, uint32_t n, int k) {
    int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    int32_t x = i % n;
    int32_t y = i / n;
    if (i < m * n) {
        dst[i] = (y + k == x);
    }
}

}  // anonymous namespace

namespace megdnn {
namespace cuda {
namespace eye {

template <typename T>
void exec_internal(T* dst, size_t m, size_t n, int k, hipStream_t stream) {
    kernel<T><<<DIVUP(m * n, NR_THREADS), NR_THREADS, 0, stream>>>(dst, m, n, k);
    after_kernel_launch();
}

#define INST(T)   template void exec_internal<T>(T*, size_t, size_t, int, hipStream_t);
#define cb(DType) INST(typename DTypeTrait<DType>::ctype)
MEGDNN_FOREACH_COMPUTING_DTYPE(cb)
cb(::megdnn::dtype::Bool)

}  // namespace eye
}  // namespace cuda
}  // namespace megdnn
// vim: syntax=cpp.doxygen foldmethod=marker foldmarker=f{{{,f}}}
